#include "hip/hip_runtime.h"
#include <THC/THC.h>

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
      i < (n);                                       \
      i += blockDim.x * gridDim.x)

// Use 1024 threads per block, which requires cuda sm_2x or above
const int CUDA_NUM_THREADS = 1024;

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// Kernel for fast unfold+copy
// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu)
__global__ void im2col_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w, const int height_col, const int width_col,
    float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize_h * ksize_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize_h; ++i) {
      for (int j = 0; j < ksize_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
          data_im[i * width + j] : 0;
        data_col += height_col * width_col;
      }
    }
  }
}

void im2col(hipStream_t stream, const float* data_im, const int channels,
    const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w, float* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h - ksize_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - ksize_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // Launch
  im2col_kernel <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>> (
      num_kernels, data_im, height, width, ksize_h, ksize_w,
      pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_col
  );
}

__global__ void col2im_kernel(const int n, const float* data_col,
    const int height, const int width, const int channels, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w, const int height_col, const int width_col,
    float* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < patch_w) ? 0 : (w - patch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < patch_h) ? 0 : (h - patch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);
    /*
       for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
       for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
    // the col location: [c * width * height + h_out, w_out]
    int c_col = c * patch_h * patch_w + (h - h_col * stride_h) * ksize + (w - w_col * stride_w);
    val += data_col[(c_col * height_col + h_col) * width_col + w_col];
    }
    }
     */
    // equivalent implementation
    int offset = (c * patch_h * patch_w + h * patch_w + w) * height_col * width_col;
    int coeff_h_col = (1 - stride_h * patch_w * height_col) * width_col;
    int coeff_w_col = (1 - stride_w * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

void col2im(hipStream_t stream, const float* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w, float* data_im) {
  int height_col = (height + 2 * pad_h - patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - patch_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_kernel <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>> (
      num_kernels, data_col, height, width, channels,
      patch_h, patch_w, pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_im
  );
}

extern "C"
void cunnrelease_SpatialConvolution(THCState *state,
    THCudaTensor *input,
    THCudaTensor *weight,
    THCudaTensor *bias,
    THCudaTensor *columns,
    THCudaTensor *ones,
    THCudaTensor *output,
    int nInputPlane, int nOutputPlane, int kW, int kH, int dW, int dH, int padding)
{
  THAssert(input->nDimension == 3 || input->nDimension == 4);// "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    THAssert(input->size[0] == nInputPlane);//, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
  } else {
    THAssert(input->size[1] == nInputPlane);//, "input channels and nInputPlane dont match");
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth + 2*padding - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padding - kH) / dH + 1;


  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        't', 'n',
        n_, m_, k_,
        1,
        THCudaTensor_data(state, ones), k_,
        THCudaTensor_data(state, bias), k_,
        0,
        THCudaTensor_data(state, output_n), n_
    );

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padding, padding, dH, dW,
      THCudaTensor_data(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = columns->size[1];
    long k = weight->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        'n', 'n',
        n, m, k,
        1,
        THCudaTensor_data(state, columns), n,
        THCudaTensor_data(state, weight), k,
        1,
        THCudaTensor_data(state, output_n), n
    );
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, output_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, output, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }
}
